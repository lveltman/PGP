
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>

void checkCudaError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "ERROR: %s: %s.\n", msg, hipGetErrorString(err));
        exit(0);
    }
}

void CustomReverse(float* vec, int size) {
    float* res = new float[size];
    for (int i = 0; i < size; ++i) {
        res[i] = vec[size - i - 1];
    }
}

__global__ void Reverse(float* res, float* vec, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;

    while (idx < size)
    {
	    res[idx] = vec[size - idx - 1];
	    idx += offset;
    }
}


int main(int argc, const char* argv[])
{
    int size;
    std::cin >> size;
    
    const int MAX = 33554432;
    const int MIN = 0;
    if (size < MIN && size > MAX)
    {
       std::cerr << "ERROR: Incorrect size!\n";
       exit(0);
    }

    float *hostVec = new float[size];

    for (int i = 0; i < size; ++i)
    {
        hostVec[i] = i;
    }
    std::cout << "vector size = " << size << std::endl;

    clock_t time;
    time = clock();
    
    CustomReverse(hostVec, size);
    time = clock() - time;
    std::cout << "CPU" << std::endl;
    std::cout << "time = " << (double)time/CLOCKS_PER_SEC << std::endl;


    float *deviceVec, *deviceRes;

    // Выделяем память для device копий
    hipMalloc((void**) &deviceVec, sizeof(float) * size);
    hipMalloc((void**) &deviceRes, sizeof(float) * size);
    // Копируем ввод на device
    hipMemcpy(deviceVec, hostVec, sizeof(float) * size, hipMemcpyHostToDevice);
     
    
	int blockCount = 32;
    int threadsCount = 32;   
    


    hipEvent_t start, end;
    hipEventCreate(&start);
    checkCudaError("hipEventCreate");
    hipEventCreate(&end);
    checkCudaError("hipEventCreate");
    hipEventRecord(start);
    checkCudaError("hipEventRecord");


    // Запускаем kernel
    Reverse<<<blockCount, threadsCount>>>(deviceRes, deviceVec, size);
    checkCudaError("Kernel invocation");
    
    hipEventRecord(end);
    checkCudaError("hipEventRecord");
    hipEventSynchronize(end);
    checkCudaError("hipEventSynchronize");
    float t;
    hipEventElapsedTime(&t, start, end);
    checkCudaError("hipEventElapsedTime");
    hipEventDestroy(start);
    checkCudaError("hipEventDestroy");
    hipEventDestroy(end);
    checkCudaError("hipEventDestroy");
    printf("GPU\n");
    printf("time = %f\n", t);
    printf("blocks = %d\n", blockCount);
    printf("threads = %d\n", threadsCount);
    hipMemcpy(hostVec, deviceRes, sizeof(float) * size, hipMemcpyDeviceToHost);
    checkCudaError("Memcpy");

    //const int accuracy = 10;
    //for (int i = 0; i < size - 1; ++i)
    //{
        //std::cout << std::scientific << std::setprecision(accuracy) << hostVec[i] << " ";
    //}
    //std::cout << std::scientific << std::setprecision(accuracy) << hostVec[size - 1];
    
    hipFree(deviceVec);
    checkCudaError("Free");
    
    hipFree(deviceRes);
    checkCudaError("Free");
    
    delete[] hostVec;

    return 0;
}